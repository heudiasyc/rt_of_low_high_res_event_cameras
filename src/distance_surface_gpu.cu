#include "hip/hip_runtime.h"
/**
 * distance_surface_gpu.cpp
 * This code corresponds to the computation of the distance surfaces from the edge images, on
 * CUDA-capable GPU.
 */

#include "rt_of_low_high_res_event_cameras/distance_surface_gpu.hpp"


/**
 * Global variables definition
 */

// A boolean to remember if the GPU memory has already been initialized
bool memory_initialized = false;

// GPU pointers for the various matrices
uint8_t *d_edges, *d_normalized_dist_surface;
int32_t *d_map_x, *d_dist_surface, *d_s, *d_t, *d_dist_surface_cols_max_value;

// Size in bytes of the edge images and the resulting distance surfaces
int edges_bytes, normalized_dist_surface_bytes;

// An int, holding the maximum value of the current distance surface (note that this is a GPU
// variable)
__device__ int32_t d_dist_surface_max_value;


/**
 * \brief GPU kernel function to compute the distance transform along the rows only, which is the
 * first step of the computation of the distance surface.
 *
 * \param d_edges Pointer to the edge image in GPU memory
 * \param d_map_x Pointer to the resulting partial distance surface on the rows, in GPU memory
 * \param nb_rows Number of rows (=height) of the input edge image
 * \param row_size Number of pixels per row (=width) of the input edge image
 */
__global__
void map_x_kernel(const uint8_t* d_edges, int32_t* d_map_x, const int nb_rows, const int row_size)
{
  // We first note the row index and the starting index of the row
  const int row_index = blockIdx.x * blockDim.x + threadIdx.x;
  const int row_first_id = row_index * row_size;

  // We then have to make sure the thread is inside the matrix bounds
  if(row_index < nb_rows) {
    // Setting the value for the first element
    if(d_edges[row_first_id]) {
      d_map_x[row_first_id] = 0;
    } else {
      // INT32_MAX is used here as a +∞
      d_map_x[row_first_id] = INT32_MAX;
    }

    // Forward pass (going to the right)
    for(int i = 1; i < row_size; ++i) {
      if(d_edges[row_first_id+i]) {
        d_map_x[row_first_id+i] = 0;
      } else {
        if(d_map_x[row_first_id+i-1] == INT32_MAX) {
          d_map_x[row_first_id+i] = INT32_MAX;
        } else {
          d_map_x[row_first_id+i] = 1 + d_map_x[row_first_id+i-1];
        }
      }
    }

    // Backward pass (going back to the left)
    for(int i = row_size-2; i >= 0; --i) {
      if(d_map_x[row_first_id+i+1] < d_map_x[row_first_id+i]) {
        d_map_x[row_first_id+i] = 1 + d_map_x[row_first_id+i+1];
      }
    }
  }
}


/**
 * \brief Computes the parabola ordinate (the F^i_y(j) function described in the paper used in the
 * distance transform algorithm, which link is given in the comment of the distance_surface_gpu
 * function)
 *
 * \param y Row of origin of the parabola
 * \param j Row for which we want to compute the ordinate
 * \param map_x_val Ordinate (value) of the parabola at its origin
 *
 * \return The parabola ordinate at the given row coordinates
 */
__device__
int32_t parab_ord(int32_t y, int32_t j, int32_t map_x_val)
{
  if(map_x_val == INT32_MAX) {
    return INT32_MAX;
  }
  return map_x_val*map_x_val + (j-y)*(j-y);
}


/**
 * \brief Computes the abscissa of the intersection of two consecutive parabolas (the Sep^i(u, v)
 * function described in the paper used in the distance transform algorithm, which link is given in
 * the comment of the distance_surface_gpu function)
 *
 * \param u The row (abscissa) of the first parabola
 * \param v The row (abscissa) of the second parabola
 * \param map_x_u_i Ordinate (value) of the first parabola at its origin (u,i)
 * \param map_x_v_i Ordinate (value) of the second parabola at its origin (v,i)
 *
 * \return The abscissa of intersection of the two parabolas
 */
__device__
int32_t parab_inter_abs(int32_t u, int32_t v, int32_t map_x_u_i, int32_t map_x_v_i)
{
  if(map_x_u_i == INT32_MAX || map_x_v_i == INT32_MAX) {
    return INT32_MAX;
  }
  return (v*v - u*u + map_x_v_i*map_x_v_i - map_x_u_i*map_x_u_i) / (2*(v-u));
}


/**
 * \brief GPU kernel function to compute the distance surface, using the partial distance transform
 * on the rows.
 *
 * \param d_map_x Pointer in GPU memory to the input partial distance surface on the rows
 * \param d_dist_surface Pointer in GPU memory to the resulting distance surface
 * \param d_s Pointer in GPU memory to a matrix of the same size as the distance surface, that will
 * be used internaly as the stack of abscissas of parabolas to use when creating the final distance
 * surface image 
 * \param d_t Pointer in GPU memory to a matrix of the same size as the distance surface, that will
 * be used internaly as the stack of abscissas at which the current parabola should not be used
 * anymore, that will be used when creating the final distance surface image
 * \param d_dist_surface_cols_max_value Pointer in GPU memory to an array with the same number of
 * elements as the number of columns (=width) of the distance surface, that will be helpful later
 * on for normalization reasons
 * \param nb_cols Number of columns (=width) of the input edge image
 * \param column_size Number of pixels per column (=height) of the input edge image
 */
__global__
void dist_surf_kernel(
  const int32_t* d_map_x, int32_t* d_dist_surface, int32_t* d_s, int32_t* d_t,
  int32_t* d_dist_surface_cols_max_value, const int nb_cols, const int column_size)
{
  // We first note the column index, its starting index, and the size of each row
  const int column_index = blockIdx.x * blockDim.x + threadIdx.x;
  const int column_first_id = column_index;
  const int row_size = nb_cols;

  // We have to make sure the thread is in the matrix bounds
  if(column_index < nb_cols) {
    // q is used as the s and t stacks index
    int q = 0;
    d_s[column_first_id] = 0;
    d_t[column_first_id] = 0;

    // For each element of the column, the best segment of parabola is searched
    // and is added to the stack (or replaces the whole stack if it is better
    // than all the other segments)
    for(int32_t j = 1; j < column_size; ++j) {
      while(
        q >= 0 && parab_ord(
          d_s[column_first_id+q*row_size], d_t[column_first_id+q*row_size],
          d_map_x[column_first_id + d_s[column_first_id+q*row_size] * row_size])
        > parab_ord(j, d_t[column_first_id+q*row_size], d_map_x[column_first_id+j*row_size]))
      {
        --q;
      }

      if(q < 0) {
        q = 0;
        d_s[column_first_id] = j;
      } else {
        int32_t parab_inter = parab_inter_abs(
          d_s[column_first_id+q*row_size], j,
          d_map_x[column_first_id + d_s[column_first_id + q*row_size] * row_size],
          d_map_x[column_first_id + j*row_size]);
        if(parab_inter != INT32_MAX) {
          int32_t w = 1 + parab_inter;
          if(w >= 0 && w < column_size) {
            ++q;
            d_s[column_first_id+q*row_size] = j;
            d_t[column_first_id+q*row_size] = w;
          }
        }
      }
    }

    // Once all the segments for the column were determined, the values are computed and attributed
    // to the cells, and the segments of hyperbolas are removed from the stack once the next best
    // segment is reached
    for(int32_t j = column_size-1; j >= 0; --j) {
      int32_t cell_value = parab_ord(
        d_s[column_first_id+q*row_size], j,
        d_map_x[column_first_id + d_s[column_first_id+q*row_size] * row_size]);
      d_dist_surface[column_first_id+j*row_size] = cell_value;
      if(j == d_t[column_first_id+q*row_size]) {
        --q;
      }

      // We also try to find here the max value in the column, and we store it for further
      // normalization purposes
      if(j == column_size-1 || cell_value > d_dist_surface_cols_max_value[column_first_id]) {
        d_dist_surface_cols_max_value[column_first_id] = cell_value;
      }
    }
  }
}


/**
 * \brief Finds the maximum value of the distance surface matrix, based on the maximum value of
 * each of its columns (computed in the dist_surf_kernel)
 *
 * \param d_dist_surface_cols_max_value Pointer in GPU memory to an array containing for each cell
 * the maximum value of the corresponding column of the distance surface computed using the
 * dist_surf_kernel function
 * \param nb_cols Number of columns of the distance surface
 */
__global__
void find_dist_surface_max_value(const int32_t* d_dist_surface_cols_max_value, const int nb_cols)
{
  int32_t max = 0;
  for(int i = 0; i < nb_cols; i++) {
    if(d_dist_surface_cols_max_value[i] > max) {
      max = d_dist_surface_cols_max_value[i];
    }
  }
  d_dist_surface_max_value = max;
}


/**
 * \brief GPU kernel function to normalize the distance surface to an uint8_t image (values between
 * 0 and 255), and to apply the correct distance surface formulation (linear, exponential, ...).
 * This is the third (and final) step of the computation of the distance surface.
 *
 * \param d_dist_surface Pointer in GPU memory to the input distance surface
 * \param d_normalized_dist_surface Pointer in GPU memory to the output normalized distance surface
 * \param nb_elems Number of pixels in the distance surface image
 * \param formulation The distance surface formulation that should be used, represented as an
 * integer
 */
__global__
void ds_norm_kernel(
  const int32_t* d_dist_surface, uint8_t* d_normalized_dist_surface, const int nb_elems,
  uint8_t formulation)
{
  // We first note the pixel index, and the stride that should be used
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  const int stride = blockDim.x * gridDim.x;

  // We then normalize the distance surface, using the correct formulation, and using the
  // previously computed maximum value of the distance surface
  if(formulation == 0) { // Linear
    for(int i = index; i < nb_elems; i += stride) {
      // The sqrt here is because the dist_surface originally contains the squared distance, but we
      // want here to have the Euclidean distance.
      // Other note: the ceil() is important here, because we don't want values close to 0 to be
      // rounded to 0. By using ceil(), we make sure than any value slightly superior to 0 will be
      // set to 1 (and the call to min() prevents any overflow, which can sometimes happen
      // strangely??)
      d_normalized_dist_surface[i] = min(
        ceil(sqrtf((float)d_dist_surface[i] / d_dist_surface_max_value) * 255), 255.);
    }
  } else if(formulation == 1) { // Linear bound
    float max = min(sqrtf(d_dist_surface_max_value), (float)DISTANCE_SURFACE_SATURATION_DISTANCE);
    for(int i = index; i < nb_elems; i += stride) {
      d_normalized_dist_surface[i] = min(
        ceil(
          min(sqrtf(d_dist_surface[i]), (float)DISTANCE_SURFACE_SATURATION_DISTANCE) / max * 255),
        255.);
    }
  } else if(formulation == 2) { // Logarithmic
    float max = logf(sqrtf(d_dist_surface_max_value)+1);
    for(int i = index; i < nb_elems; i += stride) {
      d_normalized_dist_surface[i] = min(ceil(logf(sqrtf(d_dist_surface[i])+1) / max * 255), 255.);
    }
  } else if(formulation == 3) { // Exponential
    for(int i = index; i < nb_elems; i += stride) {
      d_normalized_dist_surface[i] = min(
        (1-expf(-sqrtf(d_dist_surface[i])/ALPHA)) * 255, 255.);
    }
  } else {
    printf("Wrong distance surface formulation index!\n");
  }
}


/**
 * \brief Allocates the adequate memory on the GPU.
 *
 * \param edges An edge image, used to compute the number of bytes that should be reserved on the
 * GPU
 */
void init_cuda_memory(const Mat& edges)
{
  // The matrices that will be used are:
  // - the edge image, of type uint8_t
  // - the partial distance surface on the rows only (map_x), of type int32_t
  // - the stacks for keeping the correct parabolas, s and t, of type int32_t
  // - an array keeping the max value per column, of type int32_t
  // - the output normalized distance surface, of type uint8_t

  // Calculating the total number of bytes per matrix
  edges_bytes = edges.step * edges.rows;
  const int map_x_bytes = edges_bytes * 4;
  const int dist_surface_bytes = edges_bytes * 4;
  const int s_t_bytes = edges_bytes * 4;
  const int dist_surface_cols_max_value_bytes = edges.cols * 4;
  normalized_dist_surface_bytes = edges_bytes;

  // Allocating memory on the GPU
  hipMalloc<uint8_t>(&d_edges, edges_bytes);
  hipMalloc<int32_t>(&d_map_x, map_x_bytes);
  hipMalloc<int32_t>(&d_dist_surface, dist_surface_bytes);
  hipMalloc<int32_t>(&d_s, s_t_bytes);
  hipMalloc<int32_t>(&d_t, s_t_bytes);
  hipMalloc<int32_t>(&d_dist_surface_cols_max_value,
                      dist_surface_cols_max_value_bytes);
  hipMalloc<uint8_t>(&d_normalized_dist_surface,
                      normalized_dist_surface_bytes);
}


/**
 * \brief Frees the memory allocated on GPU.
 * It must be called manually at the end of the execution!
 */
void free_gpu_memory()
{
  // Freeing the memory allocated to all matrices if needed
  if(memory_initialized) {
    hipFree(d_edges);
    hipFree(d_map_x);
    hipFree(d_dist_surface);
    hipFree(d_s);
    hipFree(d_t);
    hipFree(d_dist_surface_cols_max_value);
    hipFree(d_normalized_dist_surface);
  }
}


/**
 * \brief Computes the distance surface from an edge image, on GPU, using the exact method
 * described in
 * https://pageperso.lif.univ-mrs.fr/~edouard.thiel/print/2007-geodis-thiel-coeurjolly.pdf
 * (the algorithm is given in the part 5.4.2), which was itself inspired by the method described in
 * the following paper: http://fab.cba.mit.edu/classes/S62.12/docs/Meijster_distance.pdf
 *
 * \param edges The input edge image
 * \param formulation The distance surface formulation that should be used
 *
 * \return The distance surface, computed from the input edge image
 */
Mat distance_surface_gpu(const Mat& edges, string formulation)
{
  // Before beginning, we must check that the given formulation for the distance surface is
  // correct; but also, since CUDA kernel function do not handle C++ strings, we convert it to an
  // integer representation
  uint8_t formulation_nbr;
  if(formulation == "linear") {
    formulation_nbr = 0;
  } else if(formulation == "linear-bound") {
    formulation_nbr = 1;
  } else if(formulation == "logarithmic") {
    formulation_nbr = 2;
  } else if(formulation == "exponential") {
    formulation_nbr = 3;
  } else {
    ROS_ERROR("Invalid distance surface formulation, exiting!");
    exit(1);
  }

  // Before beginning, we also have to make sure memory is already initialized
  if(!memory_initialized) {
    init_cuda_memory(edges);
    memory_initialized = true;
  }

  // Step 1: computing the distance values for the rows of the edge image
  // The computation for each row is done in // on the GPU

  // We first copy data from the edges Mat to device memory
  hipMemcpy(d_edges, edges.ptr(), edges_bytes, hipMemcpyHostToDevice);

  // We then launch the map_x computation kernel, as a set of blocks, each block containing
  // 64 threads
  const int threads_per_block_map_x = 64;
  const int nb_blocks_map_x = (edges.rows + threads_per_block_map_x - 1) / threads_per_block_map_x;
  map_x_kernel<<<nb_blocks_map_x, threads_per_block_map_x>>>(
    d_edges, d_map_x, edges.rows, edges.cols);

  // And we wait for the computation to finish
  hipDeviceSynchronize();

  // Step 2: computing the final distance surface using the partial one computed in step 1

  // We launch the distance surface computation kernel, as a set of blocks, each block containing
  // 64 threads
  const int threads_per_block_ds = 64;
  const int nb_blocks_ds = (edges.cols + threads_per_block_ds - 1) / threads_per_block_ds;
  dist_surf_kernel<<<nb_blocks_ds, threads_per_block_ds>>>(
    d_map_x, d_dist_surface, d_s, d_t, d_dist_surface_cols_max_value, edges.cols, edges.rows);

  // And we wait for the computation to finish
  hipDeviceSynchronize();

  // Step 3: normalizing the distance surface. This step has two purposes:
  // 1) creating an uint8_t image, that is, an image which values are integers between 0 and 255
  // 2) applying the specific required formulation for the distance surface (linear, linear-bound,
  //    logarithmic, or exponential)

  // For every formulation other than the inverse exponential one, we first have to find the
  // maximum value of the distance surface. For that purpose, we use the previously maximum values
  // found for each column.
  // Note also that a GPU kernel is used here, but without any parallelization. We could use a CPU
  // function, but since these maximum values are stored in the GPU memory, we want to avoid
  // useless copies, hence the GPU kernel.
  if(formulation_nbr != 3) {
    find_dist_surface_max_value<<<1, 1>>>(d_dist_surface_cols_max_value, edges.cols);

    // And we wait for the computation to finish
    hipDeviceSynchronize();
  }

  // We then launch the distance surface normalization kernel, as a set of blocks, each block
  // containing 64 threads
  const int threads_per_block_norm = 64;
  const int nb_blocks_norm = (edges.cols*edges.rows + threads_per_block_norm - 1)
    / threads_per_block_norm;
  ds_norm_kernel<<<nb_blocks_norm, threads_per_block_norm>>>(
    d_dist_surface, d_normalized_dist_surface, edges.cols*edges.rows, formulation_nbr);

  // And we wait once again for the computation to finish
  hipDeviceSynchronize();

  // Finally, we copy the normalized distance surface from GPU to host memory
  Mat dist_surface(edges.rows, edges.cols, CV_8U);
  hipMemcpy(
    dist_surface.ptr(), d_normalized_dist_surface, normalized_dist_surface_bytes,
    hipMemcpyDeviceToHost);

  // And we return the matrix
  return dist_surface;
}
