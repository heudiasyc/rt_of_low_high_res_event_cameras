#include "hip/hip_runtime.h"
/**
 * denoising_filling_gpu.cu
 * This code corresponds to the computation of the denoising and filling step for the edge images,
 * on CUDA-capable GPU.
 */

#include "rt_of_low_high_res_event_cameras/denoising_filling_gpu.hpp"


/**
 * Global variables definition
 */

// A boolean to remember if the GPU memory has already been initialized
bool memory_initialized = false;

// GPU pointers for the edge image, the denoised edge image, and the filled edge image respectively
uint8_t *d_edges, *d_denoised_edges, *d_filled_edges;

// Size in bytes of the edge images 
int edges_bytes;


/**
 * \brief GPU kernel function, which computes the denoised edge image.
 *
 * \param d_edges Pointer to the edge image, in GPU memory
 * \param d_denoised_edges Pointer to the denoised edge image, in GPU memory
 * \param nb_rows Number of rows of the edge image
 * \param nb_cols Number of columns of the edge image
 * \param denoising_min_neighbours Denoising threshold (N_d)
 */
__global__
void denoising_kernel(
  const uint8_t* d_edges, uint8_t* d_denoised_edges, const int nb_rows, const int nb_cols,
  const int denoising_min_neighbours)
{
  // We first compute the thread index and the stride to go through the whole edge image
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  // We then go through all the pixels asigned to this thread
  for(int i = index; i < nb_rows*nb_cols; i += stride) {
    // We apply denoising only if necessary
    if(denoising_min_neighbours > 0 && denoising_min_neighbours <= 4) {
      uint8_t val = 0;

      // If this pixel has an event (i.e. if it is an edge pixel)...
      if(d_edges[i]) {
        // ... we compute the row and column indices...
        const int row = i / nb_cols;
        const int col = i % nb_cols;

        // ... we make sure we are not too close to the border...
        if(row > 0 && row < nb_rows-1 && col > 0 && col < nb_cols-1) {
          // ... and we compute the number of direct neighbours that are also edge pixels.
          uint8_t nb_neighbours = 0;
          for(int y = -1; y <= 1; ++y) {
            for(int x = -1; x <= 1; ++x) {
              if((x == 0 || y == 0) && x != y) {
                if(d_edges[i+y*nb_cols+x]) {
                  ++nb_neighbours;
                }
              }
            }
          }

          // If there are enough correct neighbours, the edge pixel is kept
          if(nb_neighbours >= denoising_min_neighbours) {
            val = 255;
          }
        }
      }

      // We asign the final value to the pixel
      d_denoised_edges[i] = val;
    } else {
      // If denoising wasn't asked, we simply copy the data
      d_denoised_edges[i] = d_edges[i];
    }
  }
}


/**
 * \brief GPU kernel function, which computes the filled edge image.
 *
 * \param d_denoised_edges Pointer to the denoised edge image, in GPU memory
 * \param d_filled_edges Pointer to the filled edge image, in GPU memory
 * \param nb_rows Number of rows of the edge image
 * \param nb_cols Number of columns of the edge image
 * \param filling_min_neighbours Filling threshold (N_f)
 */
__global__
void filling_kernel(
  const uint8_t* d_denoised_edges, uint8_t* d_filled_edges, const int nb_rows, const int nb_cols,
  const int filling_min_neighbours)
{
  // We first compute the thread index and the stride to go through the whole denoised edge image
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  // We go through all the pixels asigned to this thread
  for(int i = index; i < nb_rows*nb_cols; i += stride) {
    // We apply filling only if necessary
    if(filling_min_neighbours > 0 && filling_min_neighbours <= 4) {
      uint8_t val = 0;

      // If this pixel has no event (i.e. if it is not an edge pixel)...
      if(!d_denoised_edges[i]) {
        // ... we compute the row and column indices...
        const int row = i / nb_cols;
        const int col = i % nb_cols;

        // ... we make sure we are not too close to the border...
        if(row > 0 && row < nb_rows-1 && col > 0 && col < nb_cols-1) {
          // ... and we compute the number of direct neighbours that are edge pixels.
          uint8_t nb_neighbours = 0;
          for(int y = -1; y <= 1; ++y) {
            for(int x = -1; x <= 1; ++x) {
              if((x == 0 || y == 0) && x != y) {
                if(d_denoised_edges[i+y*nb_cols+x]) {
                  ++nb_neighbours;
                }
              }
            }
          }

          // If there are enough correct neighbours, the pixel becomes a new edge pixel
          if(nb_neighbours >= filling_min_neighbours) {
            val = 255;
          }
        }
      } else {
        // If the pixel was already an edge pixel, then it should remain one
        val = 255;
      }

      // We asign the final value to the pixel
      d_filled_edges[i] = val;
    } else {
      // If filling wasn't asked, we simply copy the data
      d_filled_edges[i] = d_denoised_edges[i];
    }
  }
}


/**
 * \brief Allocates the adequate memory on the GPU.
 *
 * \param edges An edge image, used to compute the number of bytes that should be reserved on the
 * GPU
 */
void init_gpu_memory(const Mat& edges)
{
  // Calculating the total size in bytes of the edges matrix
  edges_bytes = edges.step * edges.rows;

  // Allocating memory on the GPU, for the three matrices (edges, denoised_edges, filled_edges)
  hipMalloc<uint8_t>(&d_edges, edges_bytes);
  hipMalloc<uint8_t>(&d_denoised_edges, edges_bytes);
  hipMalloc<uint8_t>(&d_filled_edges, edges_bytes);
}


/**
 * \brief Frees the memory allocated on GPU.
 * It must be called manually at the end of the execution!
 */
void free_gpu_memory()
{
  // Freeing the memory allocated to all the matrices if needed
  if(memory_initialized) {
    hipFree(d_edges);
    hipFree(d_denoised_edges);
    hipFree(d_filled_edges);
  }
}


/**
 * \brief Computes the denoised & filled version of the input edge image, using the GPU.
 *
 * \param edges The noisy edge image
 * \param denoising_min_neighbours Threshold for the denoising step (N_d)
 * \param filling_min_neighbours Threshold for the filling step (N_f)
 *
 * \return A denoised and filled copy of the input edge image
 */
Mat denoising_filling_gpu(
  const Mat& edges, int denoising_min_neighbours, int filling_min_neighbours)
{
  // Before doing anything, we have to make sure memory is already initialized
  if(!memory_initialized) {
    init_gpu_memory(edges);
    memory_initialized = true;
  }

  // Step 1: Denoising
  // The computation for each pixel is done in // on the GPU

  // We first copy data from the edges Mat to device memory
  hipMemcpy(d_edges, edges.ptr(), edges_bytes, hipMemcpyHostToDevice);

  // We then launch the denoising kernel, as a set of blocks, each block containing 64 threads
  const int threads_per_block_d = 64;
  const int nb_blocks_d = (edges.rows*edges.cols + threads_per_block_d - 1) / threads_per_block_d;
  denoising_kernel<<<nb_blocks_d, threads_per_block_d>>>(
    d_edges, d_denoised_edges, edges.rows, edges.cols, denoising_min_neighbours);

  // And we wait for the computation to finish
  hipDeviceSynchronize();

  // Step 2: filling
  // The computation for each pixel is also done in // on the GPU

  // No need to copy the denoised edge image back in host memory, as it can stay on the GPU to be
  // directly used for the filling step. So, we launch the filling kernel, as a set of blocks, each
  // block containing 64 threads
  const int threads_per_block_f = 64;
  const int nb_blocks_f = (edges.rows*edges.cols + threads_per_block_f - 1) / threads_per_block_f;
  filling_kernel<<<nb_blocks_f, threads_per_block_f>>>(
    d_denoised_edges, d_filled_edges, edges.rows, edges.cols, filling_min_neighbours);

  // And we wait for the computation to finish once again
  hipDeviceSynchronize();

  // We finally copy the denoised & filled edge image from GPU to host memory
  Mat filled_edges(edges.rows, edges.cols, CV_8U);
  hipMemcpy(filled_edges.ptr(), d_filled_edges, edges_bytes, hipMemcpyDeviceToHost);

  // And we return the matrix
  return filled_edges;
}
